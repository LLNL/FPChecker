#include <stdio.h>
#include <stdlib.h>
#include "dot_product.h"
#include "cuda_launch.h"

void launch()
{ 
  int n = 3;
  int nbytes = n*sizeof(double); 
  double *d_a = 0;
  hipMalloc(&d_a, nbytes);

  double *data = (double *)malloc(nbytes);
  for (int i=0; i < n; ++i)
  {
    data[i] = (double)(i+1);
  }

  hipMemcpy((void *)d_a, (void *)data, nbytes, hipMemcpyHostToDevice);

  printf("Calling kernel\n");
  dot_prod<<<16,16>>>(d_a, d_a, nbytes);
  hipDeviceSynchronize();
  printf("done\n");
}
