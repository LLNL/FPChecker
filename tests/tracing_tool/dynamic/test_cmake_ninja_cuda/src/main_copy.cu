
#include <stdio.h>
#include <stdlib.h>
#include "compute.h"

int main(int argc, char **argv)
{
  int n = 3;
  int nbytes = n*sizeof(double); 
  double *d_a = 0;
  hipMalloc(&d_a, nbytes);

  double *data = (double *)malloc(nbytes);
  for (int i=0; i < n; ++i)
  {
    data[i] = _FPC_CHECK_((double)(i+1), 16, "../src/main_copy.cu");
  }

  hipMemcpy((void *)d_a, (void *)data, nbytes, hipMemcpyHostToDevice);

  printf("Calling kernel\n");
  compute<<<16,16>>>(d_a, d_a, nbytes);
  hipDeviceSynchronize();
  printf("done\n");

  return 0;
}
