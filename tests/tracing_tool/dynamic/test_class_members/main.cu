
#include <stdio.h>
#include <stdlib.h>
#include "compute.h"
#include "circle.h"

int main(int argc, char **argv)
{
  int n = 3;
  int nbytes = n*sizeof(double); 
  double *d_a = 0;
  hipMalloc(&d_a, nbytes);

  double *data = (double *)malloc(nbytes);
  for (int i=0; i < n; ++i)
  {
    data[i] = (double)(i+1);
  }

  hipMemcpy((void *)d_a, (void *)data, nbytes, hipMemcpyHostToDevice);

  printf("Calling kernel\n");
  compute<<<16,16>>>(d_a, d_a, nbytes);
  hipDeviceSynchronize();
  printf("done\n");

  circle();

  return 0;
}
