

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void mul(float a, float b, float *res)
{
  *res = a * b;
  // NaN
  *res = (*res)-(*res) / (*res)/(*res); 
}

__global__ void dot_prod(float *x, float *y, int size)
{
  float d;
  for (int i=0; i < size; ++i)
  {
    float tmp;
    mul(x[i], y[i], &tmp);
    d += tmp;
  }

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0) {
    printf("dot: %f\n", d);
  }
}
