#include "hip/hip_runtime.h"

#include <stdio.h>

__device__ void mul(double a, double b, double *res)
{
  *res = _FPC_CHECK_D_(a * b, 6, "dot_product.cu");
  // underflow
  *res = _FPC_CHECK_D_((*res) * (1e-300 * 1e-22), 8, "dot_product.cu");
}

__global__ void dot_prod(double *x, double *y, int size)
{
  double d;
  for (int i=0; i < size; ++i)
  {
    double tmp;
    mul(x[i], y[i], &tmp);
    d += tmp;
  }

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0) {
    printf("dot: %f\n", d);
  }
}
